#include "hip/hip_runtime.h"
// �������� ������������ ��������������� ������
// ��������� �������� ���������� ����������� �� �����������
// ������ �., ����� �.
// ���������� ����������������. ������ � ���������:
// ���. � ����. - �.: ���, 1982.
// 583 �.

#define _CRT_SECURE_NO_WARNINGS
#define _SCL_SECURE_NO_WARNINGS

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/version.h>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <sstream>
#include <string>
#include <assert.h>
#include <time.h>
#include <fstream>

using namespace std;
using namespace thrust;

// Thrust is a C++ template library for CUDA based on the Standard Template Library (STL).
// Thrust allows you to implement high performance parallel applications with minimal programming effort through a high-level interface that is fully interoperable with CUDA C.
// Thrust provides a rich collection of data parallel primitives such as scan, sort, and reduce, which can be composed together to implement complex algorithms with concise, readable source code.
// By describing your computation in terms of these high-level abstractions you provide Thrust with the freedom to select the most efficient implementation automatically.
// As a result, Thrust can be utilized in rapid prototyping of CUDA applications, where programmer productivity matters most, as well as in production, where robustness and absolute performance are crucial.
// Read more at: http://docs.nvidia.com/cuda/thrust/index.html#ixzz3hymTnQwX 

double module(thrust::device_vector<double>& x);
double delta(thrust::device_vector<double>& x, thrust::device_vector<double>& y);
unsigned long total_of(thrust::device_vector<size_t>& m);
void vector_of(thrust::device_vector<unsigned>& vector, unsigned long index, thrust::device_vector<size_t>& m);
void point_of(thrust::device_vector<double>& point, thrust::device_vector<unsigned>& vector, thrust::device_vector<size_t>& m, thrust::device_vector<double>& a, thrust::device_vector<double>& b);

template <typename T>
struct inc_functor
{
	__host__ __device__ T operator()(const T& value) const
	{
		return value + 1;
	}
};

template <typename T>
struct square_functor
{
	__host__ __device__ T operator()(const T& value) const
	{
		return value * value;
	}
};

template <typename T>
struct add_functor
{
	__host__ __device__ T operator()(const T& value1, const T& value2) const
	{
		return value1 + value2;
	}
};

template <typename T>
struct sub_functor
{
	__host__ __device__ T operator()(const T& value1, const T& value2) const
	{
		return value1 - value2;
	}
};

template <typename T>
struct mul_functor
{
	__host__ __device__ T operator()(const T& value1, const T& value2) const
	{
		return value1 * value2;
	}
};

template <typename T>
struct diff_functor
{
	__host__ __device__ T operator()(const T& value1, const T& value2) const
	{
		return thrust::max(value1 - value2, value2 - value1);
	}
};

template <typename T>
struct abs_functor
{
	__host__ __device__ T operator()(const T& value) const
	{
		return thrust::max(value, -value);
	}
};

template <typename T>
struct max_functor
{
	__host__ __device__ T operator()(const T& value1, const T& value2) const
	{
		return thrust::max(value1, value2);
	}
};


enum t_ask_mode
{
	NOASK = 0,
	ASK = 1
};

enum t_trace_mode
{
	NOTRACE = 0,
	TRACE = 1
};

t_ask_mode ask_mode = NOASK;
t_trace_mode trace_mode = NOTRACE;

/////////////////////////////////////////////////////////
// ��������� ��������
static const unsigned _count = 1;
static const size_t _n = 2;
static const size_t _m[] = {20, 20};
static const double _a[] = {0, 0};
static const double _b[] = {1000, 1000};
static const double _f1[] = {0, 0, 500};
static const double _f2[] = {100, 100, 500};
static const double* _f[] = {_f1, _f2};
static const double _w1[] = {0, 0, 3040};
static const double _w2[] = {150, 180, 1800};
static const double _w3[] = {240, 200, 800};
static const double _w4[] = {260, 90, 1200};
static const double* _w[] = {_w1, _w2, _w3, _w4};
static const double _e = 1e-8;

/////////////////////////////////////////////////////////
// ���������� ������ �������
double module(thrust::device_vector<double>& x)
{
	thrust::device_vector<double> y(x.size());
	thrust::transform(x.begin(), x.end(), y.begin(), abs_functor<double>());
	return thrust::reduce(y.begin(), y.end(), 0.0, max_functor<double>());
}

/////////////////////////////////////////////////////////
// ���������� ��������� ����� ����� ��������� ���������
double delta(thrust::device_vector<double>& x, thrust::device_vector<double>& y)
{
	size_t i = thrust::min(x.size(), y.size());
	thrust::device_vector<double> diff(thrust::max(x.size(), y.size()));
	thrust::transform(x.begin(), x.end() + i, y.begin(), diff.begin(), diff_functor<double>());
	thrust::transform(x.begin() + i, x.end(), diff.begin() + i, abs_functor<double>());
	thrust::transform(y.begin() + i, y.end(), diff.begin() + i, abs_functor<double>());
	return thrust::reduce(diff.begin(), diff.end(), 0.0, max_functor<double>());
}

/////////////////////////////////////////////////////////
// ���������� ������� �������� ��������� ������� �� ������ ����
// vector - ������ �������� ��������� �������
// index - ����� ���� �������
// m - ����� ��������� �� ������� �� ���������
void vector_of(thrust::device_vector<unsigned>& vector, unsigned long index, thrust::device_vector<size_t>& m)
{
	for (size_t i = 0; i < m.size(); i++)
	{
		unsigned long m1 = 1ul + m[i];
		vector[i] = index % m1;
		index /= m1;
	}
}

/////////////////////////////////////////////////////////
// �������������� ������� �������� ��������� �������
// � ������ ��������� �����
// vector - ������ �������� ��������� �������
// m - ����� ��������� �� ������� �� ���������
// a - ������ ����������� ��������� �����
// b - ������ ������������ ��������� �����
void point_of(thrust::device_vector<double>& point, thrust::device_vector<unsigned>& vector, thrust::device_vector<size_t>& m, thrust::device_vector<double>& a, thrust::device_vector<double>& b)
{
	for (size_t i = 0; i < m.size(); i++) point[i] = (a[i] * (m[i] - vector[i]) + b[i] * vector[i]) / m[i];
}

/////////////////////////////////////////////////////////
// ���������� ����� ����� �������
// m - ����� ��������� �� ������� �� ���������
unsigned long total_of(thrust::device_vector<size_t>& m)
{
	return thrust::transform_reduce(m.begin(), m.end(), inc_functor<size_t>(), 1UL, mul_functor<unsigned long>());
}

/////////////////////////////////////////////////////////
// �������� �������������� ����� �������, �������� �������������
// x - ���������� �����
// f - ����� ����������� �������
// a - ������ ����������� ��������� �����
// b - ������ ������������ ��������� �����
bool check(thrust::device_vector<double>& x, thrust::device_vector<double>& f, thrust::device_vector<double>& a, thrust::device_vector<double>& b)
{
	size_t n = x.size();
	for (size_t i = 0; i < n; i++) if (x[i] < a[i] && x[i] < b[i]) return false;
	for (size_t i = 0; i < n; i++) if (x[i] > a[i] && x[i] > b[i]) return false;
	for (size_t i = 0; i < f.size() / (n + 1); i++)
	{
		thrust::device_vector<double> sub(x.size());
		thrust::device_vector<double> square(x.size());
		thrust::transform(x.begin(), x.end(), f.begin() + i * (n + 1), sub.begin(), diff_functor<double>());
		thrust::transform(sub.begin(), sub.end(), square.begin(), square_functor<double>());
		if (std::sqrt(thrust::reduce(square.begin(), square.end(), 0.0, add_functor<double>())) > f[i * (n + 1) + n]) return false;
	}
	return true;
}

/////////////////////////////////////////////////////////
// ������� �������
double target(thrust::device_vector<double>& x, thrust::device_vector<double>& w)
{
	size_t n = x.size();
	double s = 0;
	for (size_t i = 0; i < w.size() / (n + 1); i++)
	{
		thrust::device_vector<double> sub(x.size());
		thrust::device_vector<double> square(x.size());
		thrust::transform(x.begin(), x.end(), w.begin() + i * (n + 1), sub.begin(), sub_functor<double>());
		thrust::transform(sub.begin(), sub.end(), square.begin(), square_functor<double>());
		s += std::sqrt(thrust::reduce(square.begin(), square.end(), 0.0, add_functor<double>())) * w[i * (n + 1) + n];
	}
	return s;
}


int main(int argc, char* argv[])
{
	// http://stackoverflow.com/questions/2236197/what-is-the-easiest-way-to-initialize-a-stdvector-with-hardcoded-elements

	unsigned count = _count;
	size_t n = _n;
	double e = _e;
	thrust::host_vector<size_t> hm(_m, _m + sizeof(_m) / sizeof(_m[0]));
	thrust::host_vector<double> ha(_a, _a + sizeof(_a) / sizeof(_a[0]));
	thrust::host_vector<double> hb(_b, _b + sizeof(_b) / sizeof(_b[0]));
	thrust::host_vector<double> hf;
	thrust::host_vector<double> hw;
	for (size_t i = 0; i < sizeof(_f) / sizeof(_f[0]); i++) for (size_t j = 0; j <= n; j++) hf.push_back(_f[i][j]);
	for (size_t i = 0; i < sizeof(_w) / sizeof(_w[0]); i++) for (size_t j = 0; j <= n; j++) hw.push_back(_w[i][j]);

	char* input_file_name = NULL;
	char* output_file_name = NULL;
	char* options_file_name = NULL;

	// ��������� ��������� � ������� Windows
	// ������� setlocale() ����� ��� ���������, ������ �������� - ��� ��������� ������, � ����� ������ LC_TYPE - ����� ��������, ������ �������� � �������� ������. 
	// ������ ������� ��������� ����� ������ "Russian", ��� ��������� ������ ������� �������, ����� ����� �������� ����� ����� �� ��� � � ��.
	setlocale(LC_ALL, "");

	for (int i = 1; i < argc; i++)
	{
		if (strcmp(argv[i], "-help") == 0)
		{
			std::cout << "Usage :\t" << argv[0] << " [...] [-input <inputfile>] [-output <outputfile>]" << std::endl;
			std::cout << "�������� ������������ ��������������� ������" << std::endl;
			std::cout << "��������� �������� ���������� ����������� �� �����������" << std::endl;
			std::cout << "(�������� ������� �������� ��������� �������)" << std::endl;
			//			std::cout << "\t-n <����������� ������������>" << std::endl;
			std::cout << "\t-c <���������� ���������� ��������� ��� ������ �������>" << std::endl;
			std::cout << "\t-m <����� ��������� �� ������� �� ���������>" << std::endl;
			std::cout << "\t-a <����������� ���������� �� ������� �� ���������>" << std::endl;
			std::cout << "\t-b <������������ ���������� �� ������� �� ���������>" << std::endl;
			std::cout << "\t-e <�������� ����������>" << std::endl;
			std::cout << "\t-ask/noask" << std::endl;
			std::cout << "\t-trace/notrace" << std::endl;
		}
		else if (strcmp(argv[i], "-ask") == 0) ask_mode = ASK;
		else if (strcmp(argv[i], "-noask") == 0) ask_mode = NOASK;
		else if (strcmp(argv[i], "-trace") == 0) trace_mode = TRACE;
		else if (strcmp(argv[i], "-notrace") == 0) trace_mode = NOTRACE;
		//		else if(strcmp(argv[i],"-n")==0) n = atoi(argv[++i]);
		else if (strcmp(argv[i], "-e") == 0) e = atof(argv[++i]);
		else if (strcmp(argv[i], "-c") == 0) count = atoi(argv[++i]);
		else if (strcmp(argv[i], "-m") == 0)
		{
			std::istringstream ss(argv[++i]);
			hm.clear();
			for (size_t i = 0; i < n; i++) hm.push_back(atoi(argv[++i]));
		}
		else if (strcmp(argv[i], "-a") == 0)
		{
			ha.clear();
			for (size_t i = 0; i < n; i++) ha.push_back(atof(argv[++i]));
		}
		else if (strcmp(argv[i], "-b") == 0)
		{
			hb.clear();
			for (size_t i = 0; i < n; i++) hb.push_back(atof(argv[++i]));
		}
		else if (strcmp(argv[i], "-input") == 0) input_file_name = argv[++i];
		else if (strcmp(argv[i], "-output") == 0) output_file_name = argv[++i];
		else if (strcmp(argv[i], "-options") == 0) options_file_name = argv[++i];
	}

	if (input_file_name != NULL) freopen(input_file_name, "r",stdin);
	if (output_file_name != NULL) freopen(output_file_name, "w",stdout);

	if (options_file_name != NULL)
	{
		hf.clear();
		hw.clear();
		std::ifstream options(options_file_name);
		if (!options.is_open()) throw "Error opening file";
		std::string line;
		while (std::getline(options, line))
		{
			std::cout << line << std::endl;
			std::stringstream lineStream(line);
			std::string id;
			std::string cell;
			thrust::host_vector<double> x;
			thrust::host_vector<size_t> y;
			std::getline(lineStream, id, ' ');
			while (std::getline(lineStream, cell, ' '))
			{
				x.push_back(stod(cell));
				y.push_back(stoi(cell));
			}
			if (id[0] == 'N') n = stoi(cell);
			if (id[0] == 'E') e = stod(cell);
			if (id[0] == 'M') hm = y;
			if (id[0] == 'A') ha = x;
			if (id[0] == 'B') hb = x;
			if (id[0] == 'F') for (size_t i = 0; i < x.size(); i++) hf.push_back(x[i]);
			if (id[0] == 'W') for (size_t i = 0; i < x.size(); i++) hw.push_back(x[i]);
		}
	}

	if (ask_mode == ASK)
	{
		//  std::cout << "������� ����������� ������������:"<< std::endl; std::cin >> n;

		std::cout << "������� ����� ��������� �� ������� �� ��������� m[" << n << "]:" << std::endl;
		hm.clear();
		for (size_t i = 0; i < n; i++)
		{
			int x;
			std::cin >> x;
			hm.push_back(x);
		}

		std::cout << "������� ����������� ���������� �� ������� �� ��������� a[" << n << "]:" << std::endl;
		ha.clear();
		for (size_t i = 0; i < n; i++)
		{
			double x;
			std::cin >> x;
			ha.push_back(x);
		}

		std::cout << "������� ������������ ���������� �� ������� �� ��������� b[" << n << "]:" << std::endl;
		hb.clear();
		for (size_t i = 0; i < n; i++)
		{
			double x;
			std::cin >> x;
			hb.push_back(x);
		}

		std::cout << "������� �������� ����������:" << std::endl;
		std::cin >> e;
		std::cout << "������� ���������� ���������� ��������� ��� ������ �������:" << std::endl;
		std::cin >> count;
	}

	// Find/set the device.
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	for (int i = 0; i < device_count; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
	}

	int major = THRUST_MAJOR_VERSION;
	int minor = THRUST_MINOR_VERSION;

	std::cout << "Thrust v" << major << "." << minor << std::endl;

	for (size_t i = 0; i < hm.size(); i++) assert(hm[i]>2);

	// ��������
	clock_t time = clock();

	thrust::device_vector<size_t> m(hm);
	thrust::device_vector<double> a(ha);
	thrust::device_vector<double> b(hb);
	thrust::device_vector<double> f(hf);
	thrust::device_vector<double> w(hw);

	thrust::host_vector<double> hx(n);

	thrust::device_vector<unsigned> v(n);
	thrust::device_vector<double> t(n);
	thrust::device_vector<double> x(n);
	thrust::device_vector<double> x1(n);
	double y;

	if (trace_mode == TRACE && count == 1) std::cout << "for #1" << std::endl;
	for (unsigned s = 0; s < count; s++)
	{
		if (trace_mode == TRACE && count == 1) std::cout << "while #1" << std::endl;
		while (true)
		{
			// ������� ������ ����� � �������, �������� �������������
			unsigned long total = total_of(m);
			unsigned long index = 0;
			while (index < total)
			{
				vector_of(v, index++, m);
				point_of(x, v, m, a, b);
				if (check(x, f, a, b)) break;
			}
			if (index >= total)
			{
				for (size_t i = 0; i < n; i++) m[i] <<= 1u;
				continue;
			}
			y = target(x, w);

			if (trace_mode == TRACE && count == 1)
			{
				thrust::copy(x.begin(), x.end(), hx.begin());
				for (size_t i = 0; i < hx.size(); i++) std::cout << hx[i] << " ";
			}
			if (trace_mode == TRACE && count == 1) std::cout << "-> " << y << std::endl;

			while (index < total)
			{
				vector_of(v, index++, m);
				point_of(x1, v, m, a, b);
				if (!check(x1, f, a, b)) continue;
				double y1 = target(x1, w);
				if (y1 > y) continue;
				thrust::copy(x1.begin(), x1.end(), x.begin());
				y = y1;

				if (trace_mode == TRACE && count == 1)
				{
					thrust::host_vector<double> hx(x);
					for (size_t i = 0; i < hx.size(); i++) std::cout << hx[i] << " ";
				}
				if (trace_mode == TRACE && count == 1) std::cout << "-> " << y << std::endl;
			}
			break;
		}

		if (trace_mode == TRACE && count == 1) std::cout << "while #2" << std::endl;
		while (true)
		{
			// ������� ��������� ����� � �������, �������� �������������
			// ��������� �������� ���������� ����������� �� �����������

			thrust::copy(x.begin(), x.end(), x1.begin()); // ���������� �������� ��������� �����

			// ���� �� ����������
			if (trace_mode == TRACE && count == 1) std::cout << "for #2" << std::endl;
			for (size_t k = 0; k < n; k++)
			{
				// �������� ���������� ����������� �� �����������
				double ak = thrust::min(a[k], b[k]);
				double bk = thrust::max(a[k], b[k]);
				size_t mk = m[k];
				thrust::copy(x.begin(), x.end(), t.begin());
				while (true)
				{
					for (size_t i = 0; i <= mk; i++)
					{
						t[k] = (ak * (mk - i) + bk * i) / mk;
						if (!check(t, f, a, b)) continue;
						double yk = target(t, w);
						if (yk > y) continue;
						y = yk;
						x[k] = t[k];
						if (trace_mode == TRACE && count == 1)
						{
							thrust::copy(x.begin(), x.end(), hx.begin());
							for (size_t i = 0; i < hx.size(); i++) std::cout << hx[i] << " ";
						}
						if (trace_mode == TRACE && count == 1) std::cout << "-> " << y << std::endl;
					}
					double dd = thrust::max(ak - bk, bk - ak);
					double cc = thrust::max(thrust::max(ak, -ak), thrust::max(-bk, bk));
					if (dd <= cc * e) break;
					double xk = x[k];
					ak = thrust::max(ak, xk - dd / mk);
					bk = thrust::min(bk, xk + dd / mk);
				}
			}

			double dd = delta(x, x1);
			double cc = thrust::max(module(x), module(x1));
			if (dd <= cc * e) break;
		}
	}

	time = clock() - time;
	double seconds = ((double)time) / CLOCKS_PER_SEC / count;

	thrust::copy(x.begin(), x.end(), hx.begin());
	std::cout << "����������� ����         : " << argv[0] << std::endl;
	std::cout << "����������� ������������ : " << n << std::endl;
	std::cout << "����� ���������          : ";
	for (size_t i = 0; i < hm.size(); i++) std::cout << hm[i] << " ";
	std::cout << std::endl;
	std::cout << "����������� ����������   : ";
	for (size_t i = 0; i < ha.size(); i++) std::cout << ha[i] << " ";
	std::cout << std::endl;
	std::cout << "������������ ����������  : ";
	for (size_t i = 0; i < hb.size(); i++) std::cout << hb[i] << " ";
	std::cout << std::endl;
	std::cout << "�������� ����������      : " << e << std::endl;
	std::cout << "����� ��������           : ";
	for (size_t i = 0; i < hx.size(); i++) std::cout << hx[i] << " ";
	std::cout << std::endl;
	std::cout << "����������� ��������     : " << y << std::endl;
	std::cout << "����� ���������� (���.)  : " << seconds << std::endl;

	getchar();
	getchar();

	return 0;
}
